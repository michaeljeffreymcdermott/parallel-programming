#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void empty() {}
 
int main() {
  float ms;
  hipEvent_t start, end;     //make event vars
  hipEventCreate(&start);    //create start event
  hipEventCreate(&end);      //create end event
  hipEventRecord(start, 0);  //start recording
  empty<<< 1,1 >>> (); 
  hipEventRecord(end, 0);    //end recording
  hipEventSynchronize(end);  //sync, you have to do this!
  hipEventElapsedTime(&ms, start, end); //get elapsed time, put in timing var
  printf("GPU = %.20lf\n", ms); //print timing
