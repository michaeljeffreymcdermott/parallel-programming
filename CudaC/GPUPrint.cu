/* 
This is an good trick to see what is going on for debugging purposes. It is terribly BAD
to print from GPU kernels in anything you want to be performance oriented though. It is not a
performance oriented feature!
*/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

 
__global__ void kernel()
{
	/* 
           this just gets some kernel specific parameters
	   this is just so you can see how non-deterministic thread timing is
	*/
        int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	int tidy = threadIdx.y + blockIdx.y * blockDim.y;
 
	/* print some stuff out */
	int size = sizeof(int);
	printf("Hello, World! size=%d   tidx=%d, tidy=%d\n", size, tidx, tidy);
	return;
}
 
int main(int argc, char** argv)
{
        /*
           Keep this in mind. in Cuda 8 compute 2.0 was deprecated and it may be 
           removed by now. CDER only currently (11/2018) supports Cuda 7 so the below
           will work and may or may not warn you about this.
        */
	printf("You compile this with 'nvcc -arch sm_20 hello.cu -o hello'\n");
	printf("You need -at least- arch of sm_20 to print from kernels\n");
	dim3 dimBlock( 16, 16, 1 );
	dim3 dimGrid( 16, 16, 1 );
 
	kernel<<<dimGrid,dimBlock>>>();
	hipDeviceSynchronize();  /* you also -need- this here to print from the kernel */
	return 1;
}
